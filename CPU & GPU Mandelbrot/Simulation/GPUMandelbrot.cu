#include "hip/hip_runtime.h"
﻿#pragma once

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#include <SFML/Graphics.hpp>
#include <GPUMandelbrot.h>

dim3 BlockSize(16, 16);

//ADD HLSL SHADER LATER -> More performant as no copies back and forth between cpu and gpu

__global__ 
void MandelbrotKernelOne(int* OutputTexture, const int2 TextureSize, const double2 Offset, const double2 DrawArea, const int MaxIterations)
{   
    unsigned int PixelX = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int PixelY = blockIdx.y * blockDim.y + threadIdx.y;

    double MandelbrotCoordX = Offset.x + DrawArea.x * (((double)PixelX / (double)TextureSize.x) - 0.5);
    double MandelbrotCoordY = Offset.y + DrawArea.y * (((double)PixelY / (double)TextureSize.y) - 0.5);

    double x = 0;
    double y = 0;
    double x2 = 0;
    double y2 = 0;

    int Iteration = 0;

    while (x2 + y2 < 4. && Iteration < MaxIterations)
    {
        y = 2 * x * y + MandelbrotCoordY;
        x = x2 - y2 + MandelbrotCoordX;
        x2 = x * x;
        y2 = y * y;
        Iteration++;
    };

    if (Iteration == MaxIterations) Iteration = 64;

    OutputTexture[PixelY * TextureSize.x + PixelX] =
        ((unsigned int)((0.5f * sin(0.1f * Iteration) + 0.5f) * 255)) +
        ((unsigned int)((0.5f * sin(0.1f * Iteration + 2.094f) + 0.5f) * 255) << 8) +
        ((unsigned int)((0.5f * sin(0.1f * Iteration + 4.188f) + 0.5f) * 255) << 16) +
        ((unsigned int)255 << 24);
    
    /*
    OutputTexture[PixelY * TextureSize.x + PixelX] =
        ((unsigned int)(Iteration) +
        ((unsigned int)(Iteration) << 8) +
        ((unsigned int)(Iteration) << 16) +
        ((unsigned int)255 << 24));
    */
}

__host__
GPUColorTexture* Setup(unsigned int Width, unsigned int Height)
{
    hipSetDevice(0);

    int CorrectedWidth = (int)std::ceilf((float)Width / (float)BlockSize.x) * BlockSize.x;
    int CorrectedHeight = (int)std::ceilf((float)Height / (float)BlockSize.y) * BlockSize.y;

    GPUColorTexture* Texture = new GPUColorTexture(CorrectedWidth, CorrectedHeight);

    return Texture;
}

__host__
void RunGPUMandelbrot(GPUColorTexture* TargetTexture, const sf::Vector2d Offset, const sf::Vector2d DrawArea, const int MaxIterations)
{       
    dim3 NumBlocks(TargetTexture->Width / BlockSize.x, TargetTexture->Height / BlockSize.y);
    int2 Size { TargetTexture->Width, TargetTexture->Height };
    double2 GPUOffset { Offset.x, Offset.y };
    double2 GPUDrawArea{ DrawArea.x, DrawArea.y };

    MandelbrotKernelOne<<<NumBlocks, BlockSize>>>(TargetTexture->GPUOutputTexture, Size, GPUOffset, GPUDrawArea, MaxIterations);
    
    auto cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    }

    cudaStatus = hipMemcpy(TargetTexture->CPUTexture, TargetTexture->GPUOutputTexture, TargetTexture->Width * TargetTexture->Height * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }
}
